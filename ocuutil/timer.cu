/*
 *  Copyright 2008-2009 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "hip/hip_runtime.h"
#include "ocuutil/defines.h"
#include "ocuutil/timer.h"



namespace ocu {    
    
    

GPUTimer::GPUTimer() 
{
  e_start = new hipEvent_t;
  e_stop = new hipEvent_t;

  hipEventCreate((hipEvent_t *)e_start);  
  hipEventCreate((hipEvent_t *)e_stop); 
}

GPUTimer::~GPUTimer() 
{ 
  hipEventDestroy(*((hipEvent_t *)e_start)); 
  hipEventDestroy(*((hipEvent_t *)e_stop)); 

  delete (hipEvent_t *)e_start;
  delete (hipEvent_t *)e_stop;
}

void GPUTimer::start() { 
  hipEventRecord(*((hipEvent_t *)e_start), 0); 
}

void GPUTimer::stop()  { 
  hipEventRecord(*((hipEvent_t *)e_stop), 0); 
}

float GPUTimer::elapsed_ms()
{
    hipEventSynchronize(*((hipEvent_t *)e_stop));
    float ms;
    hipEventElapsedTime(&ms, *((hipEvent_t *)e_start), *((hipEvent_t *)e_stop));
    return ms;
}
    
} // end namespace

