/*
 *  Copyright 2008-2009 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "hip/hip_runtime.h"
#include <cstdio>
#include "ocuutil/memory.h"
#include "ocuutil/thread.h"


namespace ocu {


void *host_malloc(size_t bytes, bool pinned, bool write_combined)
{
  if (!pinned && !write_combined) {
    return malloc(bytes);
  }
  else {
    void *result;

    // always allocate portable pinned, not just pinned
    unsigned int flag = hipHostMallocPortable;
    if (write_combined)
      flag |= hipHostMallocWriteCombined;

    if (hipHostAlloc(&result, bytes, flag) != hipSuccess) {
      printf("[ERROR] host_malloc - failed with hipError_t \"%s\"\n", hipGetErrorString(hipGetLastError()));
      return 0;
    }

    return result;
  }
    
}

void host_free(void *ptr, bool pinned)
{
  if (!pinned) {
    free(ptr);
  }
  else {
    if (hipHostFree(ptr) != hipSuccess) {
      printf("[ERROR] host_free - failed on %p with hipError_t \"%s\"\n", ptr, hipGetErrorString(hipGetLastError()));
    }
  }
}



} // end namespace

