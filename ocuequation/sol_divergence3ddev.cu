#include "hip/hip_runtime.h"
/*
 *  Copyright 2008-2009 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "ocuutil/float_routines.h"
#include "ocuutil/thread.h"
#include "ocustorage/grid3dops.h"
#include "ocuequation/sol_divergence3d.h"


template<typename T>
__global__ void Sol_Divergence3DDevice_calculate_divergence(T *u, T *v, T *w, T *divergence,
  T invhx, T invhy, T invhz, 
  int xstride, int ystride,
  int nx, int ny, int nz, int blocksInY, float invBlocksInY)
{
  int blockIdxz = truncf(blockIdx.y * invBlocksInY);
  int blockIdxy = blockIdx.y - __mul24(blockIdxz,blocksInY);

  // transpose for coalescing since k is the fastest changing index 
  int k     = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
  int j     = __mul24(blockIdxy ,blockDim.y) + threadIdx.y;
  int i     = __mul24(blockIdxz ,blockDim.z) + threadIdx.z;


  if (i < nx && j < ny && k < nz) {
    int idx = __mul24(i, xstride) + __mul24(j,ystride) + k;

    divergence[idx] = invhx * (u[idx + xstride] - u[idx]) +
                      invhy * (v[idx + ystride] - v[idx]) +
                      invhz * (w[idx + 1      ] - w[idx]);
  }
}

namespace ocu {

template<typename T>
Sol_Divergence3DDevice<T>::Sol_Divergence3DDevice()
{
  _nx = 0;
  _ny = 0;
  _nz = 0;
  _hx = 0;
  _hy = 0;
  _hz = 0;

  u = v = w = 0;
  divergence = 0;
}

template<typename T>
bool
Sol_Divergence3DDevice<T>::solve()
{
  int tnx = _nz;
  int tny = _ny;
  int tnz = _nx;

  int threadsInX = 16;
  int threadsInY = 4;
  int threadsInZ = 4;

  int blocksInX = (tnx+threadsInX-1)/threadsInX;
  int blocksInY = (tny+threadsInY-1)/threadsInY;
  int blocksInZ = (tnz+threadsInZ-1)/threadsInZ;

  dim3 Dg = dim3(blocksInX, blocksInY*blocksInZ);
  dim3 Db = dim3(threadsInX, threadsInY, threadsInZ);

  PreKernel();
  Sol_Divergence3DDevice_calculate_divergence<<<Dg, Db, 0, ThreadManager::get_compute_stream()>>>(&u->at(0,0,0),&v->at(0,0,0),&w->at(0,0,0), &divergence->at(0,0,0),
    (T)(1/_hx), (T)(1/_hy), (T)(1/_hz), 
    u->xstride(), u->ystride(), 
    _nx, _ny, _nz, blocksInY, 1.0f / (float)blocksInY);
  return PostKernel("Sol_Divergence3DDevice_calculate_divergence::calculate_divergence");
}

template<typename T>
bool 
Sol_Divergence3DDevice<T>::initialize_storage(
  int nx, int ny, int nz, double hx, double hy, double hz, Grid3DDevice<T> *u_val, Grid3DDevice<T> *v_val, Grid3DDevice<T> *w_val, Grid3DDevice<T> *divergence_val)
{
  u = u_val;
  v = v_val;
  w = w_val;
  divergence = divergence_val;

  if (!check_valid_mac_dimensions(*u_val, *v_val, *w_val, nx, ny, nz)) {
    printf("[ERROR] Sol_Divergence3DDevice::initialize_storage - u,v,w grid dimensions mismatch\n");
    return false;
  }

  if (divergence_val->nx() != nx || divergence_val->ny() != ny || divergence_val->nz() != nz ||
    !divergence_val->check_layout_match(*u_val)) {
    printf("[ERROR] Sol_Divergence3DDevice::initialize_storage - invalid dimensions for divergence\n");
    return false;
  }

  // since they all have the same layout, we only need to test u
  if (u_val->gx() < 1 || u_val->gy() < 1 || u_val->gz() < 1) {
    printf("[ERROR] Sol_Divergence3DDevice::initialize_storage - must have at least one ghost cell on all sides\n");
    return false;
  }

  if (!check_float(hx) || !check_float(hy) || !check_float(hz)) {
    printf("[ERROR] Sol_Divergence3DDevice::initialize_storage - garbage hx,hy,hz value\n");
    return false;
  }

  _hx = hx;
  _hy = hy;
  _hz = hz;

  _nx = nx;
  _ny = ny;
  _nz = nz;

  return true;
}




template class Sol_Divergence3DDevice<float>;
#ifdef OCU_DOUBLESUPPORT
template class Sol_Divergence3DDevice<double>;
#endif

} // end namespace

